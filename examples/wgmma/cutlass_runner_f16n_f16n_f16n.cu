#include <iostream>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help;
  bool error;

  int m, n, k, l, iterations;
  float alpha, beta;

  Options():
    help(false),
    error(false),
    iterations(0),
    m(2048), n(2048), k(2048), l(1),
    alpha(1.f), beta(0.f)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, 2048);
    cmd.get_cmd_line_argument("n", n, 2048);
    cmd.get_cmd_line_argument("k", k, 2048);
    cmd.get_cmd_line_argument("l", l, 1);
    cmd.get_cmd_line_argument("iterations", iterations, 0);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "runner\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent (batch count) of the GEMM\n"
      << "  --iterations=<f32>          Iterations for benchmark\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n";

    return out;
  }

  double gflops(double runtime_s) const {
    uint64_t flop;
    // Two flops per multiply-add
    if(beta == 0.f) flop = uint64_t(2) * m * n * k;
    // Three flops per multiply-add
    else flop = uint64_t(3) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result {
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(double avg_runtime_ms = 0, double gflops = 0,
         cutlass::Status status = cutlass::Status::kSuccess,
         hipError_t error = hipSuccess)
      : avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status),
        error(error), passed(false) {}
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

// Wrapper to construct, run, and verify a GEMM. This example showcases CUTLASS's collective
// operation builders by specializing the GEMM only on the kernel schedule it will use and the
// number of pipeline stages.
//
// One can use a special `Auto` type that tells the CollectiveBuilder
// to select an appropriate value on its own. The CollectiveBuilder will attempt to select
// configurations that will result in the most-performant kernel, but this is not a guarantee.
//
// If relying on 'Auto' schedules, all builders must use the 'Auto' schedule to ensure compatiblity.
// For example, if `KernelScheduleAuto` is used for the mainloop builder, `EpilogueScheduleAuto` must
// be used for the epilogue builder.
//
// Furthermore, if an override schedule is selected, both epilgoue and mainloop schedules must
// be specifically opt into a compatible selection.
//
// Behavior of the CollectiveBuilder with `Auto` types is subject to change in future releases
// -- do not rely on `Auto` if you require a specific scheduling policy.
template <
  // Type of kernel schedule to generate
  class MainloopScheduleType = cutlass::gemm::collective::KernelScheduleAuto,
  // Type of epilogue schedule to generate
  class EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto,
  
  class TileShape_MNK = Shape<_64,_128,_64>,

  class ClusterShape_MNK = Shape<_1,_1,_1>,
  // Number of pipeline stages to use
  class StageCountType = cutlass::gemm::collective::StageCountAuto
>
struct ExampleRunner {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::RowMajor;
  using LayoutC = cutlass::layout::RowMajor;

  using ElementA = cutlass::half_t;
  using ElementB = cutlass::half_t;
  using ElementAccumulator = cutlass::half_t;

  static constexpr int AlignmentA = 8;
  static constexpr int AlignmentB = 8;
  static constexpr int AlignmentD = 8;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, AlignmentA,
      ElementB, LayoutB, AlignmentB,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      MainloopScheduleType
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementAccumulator,
      cutlass::half_t, LayoutC, AlignmentD,
      cutlass::half_t, LayoutC, AlignmentD,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using ProblemShapeType = typename Gemm::GemmKernel::ProblemShape;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using LayoutTagA = cutlass::gemm::detail::StrideToLayoutTagA_t<StrideA>;
  using LayoutTagB = cutlass::gemm::detail::StrideToLayoutTagB_t<StrideB>;
  using LayoutTagC = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideC>;
  using LayoutTagD = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideD>;

  //
  // Data members
  //

  /// Initialization
  StrideA stride_A;
  StrideB stride_B;
  StrideC stride_C;
  StrideD stride_D;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
  cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
  cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
  cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

  //
  // Methods
  //

  bool verify(const ProblemShapeType& problem_size, float alpha, float beta) {
    auto [M, N, K, L] = problem_size;

    cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({M, K}));
    cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({K, N}));
    cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({M, N}));
    cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({M, N}));

    cutlass::reference::device::GemmComplex(
          {M, N, K},
          typename Gemm::EpilogueOutputOp::ElementCompute(alpha),
          ref_A,
          cutlass::ComplexTransform::kNone,
          ref_B,
          cutlass::ComplexTransform::kNone,
          typename Gemm::EpilogueOutputOp::ElementCompute(beta),
          ref_C,
          ref_D,
          typename Gemm::EpilogueOutputOp::ElementAccumulator(0.f),
          L,     // batch_count
          M * K, // batch_stride_A
          K * N, // batch_stride_B
          M * N, // batch_stride_C
          M * N  // batch_stride_D
        );

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

    return passed;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto problem_shape_MNKL = cute::append<4>(problem_size, 1);
    auto [M, N, K, L] = problem_shape_MNKL;

    stride_A = make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
    stride_B = make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
    stride_C = make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
    stride_D = make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

    block_A.reset(M * K * L);
    block_B.reset(K * N * L);
    block_C.reset(M * N * L);
    block_D.reset(M * N * L);
    block_ref_D.reset(M * N * L);

    initialize_block(block_A, seed + 2023);
    initialize_block(block_B, seed + 2022);
    initialize_block(block_C, seed + 2021);
  }

  bool run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.m, options.n, options.k, options.l};

    initialize(problem_size);
    
    ElementAccumulator alpha = ElementAccumulator(options.alpha);
    ElementAccumulator beta = ElementAccumulator(options.beta);

    typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      problem_size,
      {block_A.get(), stride_A, block_B.get(), stride_B},
      {{alpha, beta}, block_C.get(), stride_C, block_D.get(), stride_D},
      hw_info
    };

    Gemm gemm_op;

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    // Run the GEMM
    status = gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Verify that the result is correct
    bool passed = verify(problem_size, options.alpha, options.beta);
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
    }

    // Benchmark here
    if(passed && options.iterations > 0) {
      Result prof;
      GpuTimer timer;
      timer.start();
      for (int iter = 0; iter < options.iterations; ++iter)
          gemm_op.run();
      timer.stop();
      // Compute average runtime and GFLOPs.
      float elapsed_ms = timer.elapsed_millis();
      prof.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
      prof.gflops = options.gflops(prof.avg_runtime_ms / 1000.0);
      std::cout << "  Avg runtime: " << prof.avg_runtime_ms << " ms\t"
                << "  GFLOPS: " << prof.gflops << std::endl;
    }

    return passed;
  }

};

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, bool passed) {
  std::cout << description << ": " << (passed ? "Passed" : "Failed") << "\n\n";
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class ctaShape, class clusterShape>
void quickRunner(Options options, cutlass::KernelHardwareInfo hw_info) {
  bool passed;
#ifdef KERNEL_MULTISTAGE
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  // KernelMultistage 
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  // KernelMultistage + NoSmemWarpSpecialized
  ExampleRunner<cutlass::gemm::KernelMultistage, cutlass::epilogue::NoSmemWarpSpecialized, ctaShape, clusterShape> kernelmultistage;
  passed = kernelmultistage.run(options, hw_info);
  print_result("KernelMultistage + NoSmemWarpSpecialized + Auto State", passed);
#elif KERNEL_TMA
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  // KernelTma
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  // KernelTma + TmaWarpSpecialized
  ExampleRunner<cutlass::gemm::KernelTma, cutlass::epilogue::NoSmemWarpSpecialized, ctaShape, clusterShape> KernelTma1;
  passed = KernelTma1.run(options, hw_info);
  print_result("KernelTma + NoSmemWarpSpecialized + Auto State", passed);
#elif KERNEL_WARPSPECIALIZED
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  // KernelTmaWarpSpecialized
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized, cutlass::epilogue::NoSmemWarpSpecialized, ctaShape, clusterShape> ws_schedule_auto_stage_runner;
  passed = ws_schedule_auto_stage_runner.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized + NoSmemWarpSpecialized + Auto State", passed);
#elif KERNEL_WARPSPECIALIZED_EPITMA
  // KernelTmaWarpSpecialized + TmaWarpSpecialized
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized, cutlass::epilogue::TmaWarpSpecialized, ctaShape, clusterShape> KernelTmaWarpSpecialized2;
  passed = KernelTmaWarpSpecialized2.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized + TmaWarpSpecialized + Auto State", passed);
#elif KERNEL_PINGPONG
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  // KernelTmaWarpSpecializedPingpong
  ///////////////////////////////////////////////////////////////////////////////////////////////////
  ExampleRunner<
    cutlass::gemm::KernelTmaWarpSpecializedPingpong,
    cutlass::epilogue::TmaWarpSpecialized, ctaShape, clusterShape> ws_pingpong_schedule_auto_stage_runner;
  passed = ws_pingpong_schedule_auto_stage_runner.run(options, hw_info);
  print_result("KernelTmaWarpSpecializedPingpong + TmaWarpSpecialized + Auto State", passed);
#elif KERNEL_PINGPONG_EPITMA
  // KernelTmaWarpSpecialized + TmaWarpSpecialized
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecializedPingpong, cutlass::epilogue::TmaWarpSpecialized, ctaShape, clusterShape> KernelTmaWarpSpecializedPingpong2;
  passed = KernelTmaWarpSpecializedPingpong2.run(options, hw_info);
  print_result("KernelTmaWarpSpecializedPingpong + TmaWarpSpecialized + Auto State", passed);
#endif
}


int main(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major < 9) {
    std::cout
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater) and CUDA 12.0 or greater.\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  using TileShape_MNK = Shape<_128, _128, _64>;

  std::cout << "Shape<_1,_1,_1>." << std::endl;
  quickRunner<TileShape_MNK, Shape<_1,_1,_1>>(options, hw_info);

#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
